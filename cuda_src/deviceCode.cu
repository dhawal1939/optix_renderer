#include "hip/hip_runtime.h"
// // ======================================================================== //
// // Copyright 2019-2020 Ingo Wald                                            //
// //                                                                          //
// // Licensed under the Apache License, Version 2.0 (the "License");          //
// // you may not use this file except in compliance with the License.         //
// // You may obtain a copy of the License at                                  //
// //                                                                          //
// //     http://www.apache.org/licenses/LICENSE-2.0                           //
// //                                                                          //
// // Unless required by applicable law or agreed to in writing, software      //
// // distributed under the License is distributed on an "AS IS" BASIS,        //
// // WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// // See the License for the specific language governing permissions and      //
// // limitations under the License.                                           //
// // ======================================================================== //

#include <utils.cuh>
#include <common.cuh>
#include <frostbite.cuh>
#include <lcg_random.cuh>
#include <optix_device.h>
#include <owl/owl_device.h>
#include <ltc/ltc_utils.cuh>
#include <ltc/polygon_utils.cuh>

#include <hip/hip_vector_types.h>
#include <>




OPTIX_CLOSEST_HIT_PROGRAM(triangleMeshCHShadow)()
{
    const TriangleMeshData& self = owl::getProgramData<TriangleMeshData>();
    const owl::common::vec3i primitiveIndices = self.index[optixGetPrimitiveIndex()];
    ShadowRayData& srd = owl::getPRD<ShadowRayData>();

    if (self.isLight) {
        srd.visibility = owl::common::vec3f(1.f);
        srd.point = barycentricInterpolate(self.vertex, primitiveIndices);
        srd.normal = normalize(barycentricInterpolate(self.normal, primitiveIndices));
        srd.emit = self.emit;

        owl::common::vec3f v1 = self.vertex[primitiveIndices.x];
        owl::common::vec3f v2 = self.vertex[primitiveIndices.y];
        owl::common::vec3f v3 = self.vertex[primitiveIndices.z];
        srd.area = 0.5f * length(cross(v1 - v2, v3 - v2));

        srd.cg = (v1 + v2 + v3) / 3.f;
    }
    else {
        srd.visibility = owl::common::vec3f(0.f);
        srd.point = barycentricInterpolate(self.vertex, primitiveIndices);
        srd.normal = normalize(barycentricInterpolate(self.normal, primitiveIndices));
        srd.emit = owl::common::vec3f(0.);

        owl::common::vec3f v1 = self.vertex[primitiveIndices.x];
        owl::common::vec3f v2 = self.vertex[primitiveIndices.y];
        owl::common::vec3f v3 = self.vertex[primitiveIndices.z];
        srd.area = 0.5f * length(cross(v1 - v2, v3 - v2));

        srd.cg = (v1 + v2 + v3) / 3.f;
    }

}

OPTIX_CLOSEST_HIT_PROGRAM(triangleMeshCH)()
{
    const TriangleMeshData& self = owl::getProgramData<TriangleMeshData>();
    const owl::common::vec3i primitiveIndices = self.index[optixGetPrimitiveIndex()];

    SurfaceInteraction& si = owl::getPRD<SurfaceInteraction>();

    // Exact hit point on the triangle
    si.p = barycentricInterpolate(self.vertex, primitiveIndices);

    // Out going direction pointing toward the pixel location
    si.wo = owl::normalize(optixLaunchParams.camera.pos - si.p);

    // UV coordinate of the hit point
    si.uv = barycentricInterpolate(self.texCoord, primitiveIndices);

    // geometric normal 
    si.n_geom = normalize(barycentricInterpolate(self.normal, primitiveIndices));

    // Initializes to_local from n_geo then obtains to_world by taking inverse of the to_local
    orthonormalBasis(si.n_geom, si.to_local, si.to_world);

    // obtain wo is in world space cam_pos - hit_loc_world get local frame of the wo as wo_local
    si.wo_local = normalize(apply_mat(si.to_local, si.wo));

    // axix independet prop
    si.diffuse = self.diffuse;
    if (self.hasDiffuseTexture)
        si.diffuse = (owl::common::vec3f)tex2D<float4>(self.diffuse_texture, si.uv.x, si.uv.y);

    si.alpha = self.alpha;
    if (self.hasAlphaTexture)
        si.alpha = tex2D<float4>(self.alpha_texture, si.uv.x, si.uv.y).x;
    si.alpha = clamp(si.alpha, 0.01f, 1.f);

    si.emit = self.emit;
    si.isLight = self.isLight;

    si.hit = true;
}

OPTIX_MISS_PROGRAM(miss)()
{
    const owl::common::vec2i pixelId = owl::getLaunchIndex();
    const MissProgData& self = owl::getProgramData<MissProgData>();

    SurfaceInteraction& si = owl::getPRD<SurfaceInteraction>();
    si.hit = false;
    si.diffuse = self.const_color;
}


__device__
owl::common::vec3f integrateOverPolygon(SurfaceInteraction& si, owl::common::vec3f ltc_mat[3], owl::common::vec3f ltc_mat_inv[3], float amplitude,
    owl::common::vec3f iso_frame[3], TriLight& triLight)
{
    owl::common::vec3f lv1 = triLight.v1;
    owl::common::vec3f lv2 = triLight.v2;
    owl::common::vec3f lv3 = triLight.v3;
    owl::common::vec3f lemit = triLight.emit;
    owl::common::vec3f lnormal = triLight.normal;

    // Move to origin and normalize
    lv1 = owl::normalize(lv1 - si.p);
    lv2 = owl::normalize(lv2 - si.p);
    lv3 = owl::normalize(lv3 - si.p);

    owl::common::vec3f cg = normalize(lv1 + lv2 + lv3);
    if (owl::dot(-cg, lnormal) < 0.f)
        return owl::common::vec3f(0.f);

    lv1 = owl::normalize(apply_mat(si.to_local, lv1));
    lv2 = owl::normalize(apply_mat(si.to_local, lv2));
    lv3 = owl::normalize(apply_mat(si.to_local, lv3));

    lv1 = owl::normalize(apply_mat(iso_frame, lv1));
    lv2 = owl::normalize(apply_mat(iso_frame, lv2));
    lv3 = owl::normalize(apply_mat(iso_frame, lv3));

    float diffuse_shading = 0.f;
    float ggx_shading = 0.f;

    owl::common::vec3f diff_clipped[5] = { lv1, lv2, lv3, lv1, lv1 };
    int diff_vcount = clipPolygon(3, diff_clipped);

    if (diff_vcount == 3) {
        diffuse_shading = integrateEdge(diff_clipped[0], diff_clipped[1]);
        diffuse_shading += integrateEdge(diff_clipped[1], diff_clipped[2]);
        diffuse_shading += integrateEdge(diff_clipped[2], diff_clipped[0]);
        diffuse_shading = owl::abs(diffuse_shading);
    }
    else if (diff_vcount == 4) {
        diffuse_shading = integrateEdge(diff_clipped[0], diff_clipped[1]);
        diffuse_shading += integrateEdge(diff_clipped[1], diff_clipped[2]);
        diffuse_shading += integrateEdge(diff_clipped[2], diff_clipped[3]);
        diffuse_shading += integrateEdge(diff_clipped[3], diff_clipped[0]);
        diffuse_shading = owl::abs(diffuse_shading);
    }

    diff_clipped[0] = owl::normalize(apply_mat(ltc_mat_inv, lv1));
    diff_clipped[1] = owl::normalize(apply_mat(ltc_mat_inv, lv2));
    diff_clipped[2] = owl::normalize(apply_mat(ltc_mat_inv, lv3));
    diff_clipped[3] = owl::normalize(apply_mat(ltc_mat_inv, lv1));
    diff_clipped[4] = owl::normalize(apply_mat(ltc_mat_inv, lv1));

    owl::common::vec3f ltc_clipped[5] = { diff_clipped[0], diff_clipped[1], diff_clipped[2], diff_clipped[3], diff_clipped[4] };
    int ltc_vcount = clipPolygon(diff_vcount, ltc_clipped);

    if (ltc_vcount == 3) {
        ggx_shading = integrateEdge(ltc_clipped[0], ltc_clipped[1]);
        ggx_shading += integrateEdge(ltc_clipped[1], ltc_clipped[2]);
        ggx_shading += integrateEdge(ltc_clipped[2], ltc_clipped[0]);
        ggx_shading = owl::abs(ggx_shading);
    }
    else if (ltc_vcount == 4) {
        ggx_shading = integrateEdge(ltc_clipped[0], ltc_clipped[1]);
        ggx_shading += integrateEdge(ltc_clipped[1], ltc_clipped[2]);
        ggx_shading += integrateEdge(ltc_clipped[2], ltc_clipped[3]);
        ggx_shading += integrateEdge(ltc_clipped[3], ltc_clipped[0]);
        ggx_shading = owl::abs(ggx_shading);
    }
    else if (ltc_vcount == 5) {
        ggx_shading = integrateEdge(ltc_clipped[0], ltc_clipped[1]);
        ggx_shading += integrateEdge(ltc_clipped[1], ltc_clipped[2]);
        ggx_shading += integrateEdge(ltc_clipped[2], ltc_clipped[3]);
        ggx_shading += integrateEdge(ltc_clipped[3], ltc_clipped[4]);
        ggx_shading += integrateEdge(ltc_clipped[4], ltc_clipped[0]);
        ggx_shading = owl::abs(ggx_shading);
    }

    owl::common::vec3f color = (si.diffuse * lemit * diffuse_shading) + (amplitude * lemit * ggx_shading);
    return color;
}

__device__
owl::common::vec3f sampleLightSource(SurfaceInteraction si, int lightIdx, float lightSelectionPdf, owl::common::vec2f rand, bool mis)
{
    owl::common::vec3f color(0.f, 0.f, 0.f);
    float light_pdf = 0.f, brdf_pdf = 0.f;
    TriLight triLight = optixLaunchParams.triLights[lightIdx];

    owl::common::vec3f lv1 = triLight.v1;
    owl::common::vec3f lv2 = triLight.v2;
    owl::common::vec3f lv3 = triLight.v3;
    owl::common::vec3f lnormal = triLight.normal;
    owl::common::vec3f lemit = triLight.emit;
    float larea = triLight.area;

    owl::common::vec3f lpoint = samplePointOnTriangle(lv1, lv2, lv3, rand.x, rand.y);
    owl::common::vec3f wi = normalize(lpoint - si.p);
    owl::common::vec3f wi_local = normalize(apply_mat(si.to_local, wi));

    float xmy = pow(owl::length(lpoint - si.p), 2.f);
    float lDotWi = owl::abs(owl::dot(lnormal, -wi));

    light_pdf = lightSelectionPdf * (xmy / (larea * lDotWi));

    ShadowRay ray;
    ray.origin = si.p + 1e-3f * si.n_geom;
    ray.direction = wi;

    ShadowRayData srd;
    owl::traceRay(optixLaunchParams.world, ray, srd);

    if (si.wo_local.z > 0.f && wi_local.z > 0.f && srd.visibility != owl::common::vec3f(0.f) && light_pdf > 0.f && owl::dot(-wi, lnormal) > 0.f) {
        owl::common::vec3f brdf = evaluate_brdf(si.wo_local, wi_local, si.diffuse, si.alpha);
        brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + wi_local));

        if (mis && brdf_pdf > 0.f) {
            float weight = PowerHeuristic(1, light_pdf, 1, brdf_pdf);
            color += brdf * lemit * owl::abs(wi_local.z) * weight / light_pdf;
        }
        else if (!mis) {
            color += brdf * lemit * owl::abs(wi_local.z) / light_pdf;
        }
    }

    return color;
}

__device__
owl::common::vec3f sampleLightSourceNoNLTest(SurfaceInteraction si, int lightIdx, float lightSelectionPdf, owl::common::vec2f rand, bool mis)
{
    owl::common::vec3f color(0.f, 0.f, 0.f);
    float light_pdf = 0.f, brdf_pdf = 0.f;
    TriLight triLight = optixLaunchParams.triLights[lightIdx];

    owl::common::vec3f lv1 = triLight.v1;
    owl::common::vec3f lv2 = triLight.v2;
    owl::common::vec3f lv3 = triLight.v3;
    owl::common::vec3f lnormal = triLight.normal;
    owl::common::vec3f lemit = triLight.emit;
    float larea = triLight.area;

    owl::common::vec3f lpoint = samplePointOnTriangle(lv1, lv2, lv3, rand.x, rand.y);
    owl::common::vec3f wi = normalize(lpoint - si.p);
    owl::common::vec3f wi_local = normalize(apply_mat(si.to_local, wi));

    float xmy = pow(owl::length(lpoint - si.p), 2.f);
    float lDotWi = owl::abs(owl::dot(lnormal, -wi));

    light_pdf = lightSelectionPdf * (xmy / (larea * lDotWi));

    ShadowRay ray;
    ray.origin = si.p + 1e-3f * si.n_geom;
    ray.direction = wi;

    ShadowRayData srd;
    owl::traceRay(optixLaunchParams.world, ray, srd);

    if (si.wo_local.z > 0.f && wi_local.z > 0.f && srd.visibility != owl::common::vec3f(0.f) && light_pdf > 0.f) {
        owl::common::vec3f brdf = evaluate_brdf(si.wo_local, wi_local, si.diffuse, si.alpha);
        brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + wi_local));

        if (mis && brdf_pdf > 0.f) {
            float weight = PowerHeuristic(1, light_pdf, 1, brdf_pdf);
            color += brdf * lemit * owl::abs(wi_local.z) * weight / light_pdf;
        }
        else if (!mis) {
            color += brdf * lemit * owl::abs(wi_local.z) / light_pdf;
        }
    }

    return color;
}

__device__
owl::common::vec3f sampleLightSourceNoVis(SurfaceInteraction si, int lightIdx, float lightSelectionPdf, owl::common::vec2f rand, bool mis)
{
    owl::common::vec3f color(0.f, 0.f, 0.f);
    float light_pdf = 0.f, brdf_pdf = 0.f;
    TriLight triLight = optixLaunchParams.triLights[lightIdx];

    owl::common::vec3f lv1 = triLight.v1;
    owl::common::vec3f lv2 = triLight.v2;
    owl::common::vec3f lv3 = triLight.v3;
    owl::common::vec3f lnormal = triLight.normal;
    owl::common::vec3f lemit = triLight.emit;
    float larea = triLight.area;

    owl::common::vec3f lpoint = samplePointOnTriangle(lv1, lv2, lv3, rand.x, rand.y);
    owl::common::vec3f wi = normalize(lpoint - si.p);
    owl::common::vec3f wi_local = normalize(apply_mat(si.to_local, wi));

    float xmy = pow(owl::length(lpoint - si.p), 2.f);
    float lDotWi = owl::abs(owl::dot(lnormal, -wi));

    light_pdf = lightSelectionPdf * (xmy / (larea * lDotWi));

    ShadowRay ray;
    ray.origin = si.p + 1e-3f * si.n_geom;
    ray.direction = wi;

    ShadowRayData srd;
    owl::traceRay(optixLaunchParams.world, ray, srd);

    if (si.wo_local.z > 0.f && wi_local.z > 0.f && light_pdf > 0.f) {
        owl::common::vec3f brdf = evaluate_brdf(si.wo_local, wi_local, si.diffuse, si.alpha);
        brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + wi_local));

        if (mis && brdf_pdf > 0.f) {
            float weight = PowerHeuristic(1, light_pdf, 1, brdf_pdf);
            color += brdf * lemit * owl::abs(wi_local.z) * weight / light_pdf;
        }
        else if (!mis) {
            color += brdf * lemit * owl::abs(wi_local.z) / light_pdf;
        }
    }

    return color;
}

__device__
owl::common::vec3f sampleBRDF(SurfaceInteraction si, float lightSelectionPdf, owl::common::vec2f rand, bool mis)
{
    owl::common::vec3f wi_local = sample_GGX(rand, si.alpha, si.wo_local);
    owl::common::vec3f wi = normalize(apply_mat(si.to_world, wi_local));

    ShadowRay ray;
    ShadowRayData srd;
    ray.origin = si.p + 1e-3f * si.n_geom;
    ray.direction = wi;
    owl::traceRay(optixLaunchParams.world, ray, srd);

    owl::common::vec3f color(0.f, 0.f, 0.f);
    float light_pdf = 0.f, brdf_pdf = 0.f;

    if (wi_local.z > 0.f && si.wo_local.z > 0.f && srd.visibility != owl::common::vec3f(0.f)) {
        float xmy = pow(owl::length(srd.point - si.p), 2.f);
        float lDotWi = owl::abs(owl::dot(srd.normal, -wi));
        light_pdf = lightSelectionPdf * (xmy / (srd.area * lDotWi));

        owl::common::vec3f brdf = evaluate_brdf(si.wo_local, wi_local, si.diffuse, si.alpha);
        brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + wi_local));

        if (mis && light_pdf > 0.f && brdf_pdf > 0.f) {
            float weight = PowerHeuristic(1, brdf_pdf, 1, light_pdf);
            color += brdf * srd.emit * owl::abs(wi_local.z) * weight / brdf_pdf;
        }
        else if (!mis && brdf_pdf > 0.f) {
            color += brdf * srd.emit * owl::abs(wi_local.z) / brdf_pdf;
        }
    }

    return color;
}

__device__
owl::common::vec3f sampleBRDFNoNLTest(SurfaceInteraction si, float lightSelectionPdf, owl::common::vec2f rand, bool mis)
{
    owl::common::vec3f wi_local = sample_GGX(rand, si.alpha, si.wo_local);
    owl::common::vec3f wi = normalize(apply_mat(si.to_world, wi_local));

    ShadowRay ray;
    ShadowRayData srd;
    ray.origin = si.p + 1e-3f * si.n_geom;
    ray.direction = wi;
    owl::traceRay(optixLaunchParams.world, ray, srd);

    owl::common::vec3f color(0.f, 0.f, 0.f);
    float light_pdf = 0.f, brdf_pdf = 0.f;

    if (srd.visibility != owl::common::vec3f(0.f)) {
        float xmy = pow(owl::length(srd.point - si.p), 2.f);
        float lDotWi = owl::abs(owl::dot(srd.normal, -wi));
        light_pdf = lightSelectionPdf * (xmy / (srd.area * lDotWi));

        owl::common::vec3f brdf = evaluate_brdf(si.wo_local, wi_local, si.diffuse, si.alpha);
        brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + wi_local));

        if (mis && light_pdf > 0.f && brdf_pdf > 0.f) {
            float weight = PowerHeuristic(1, brdf_pdf, 1, light_pdf);
            color += brdf * srd.emit * owl::abs(wi_local.z) * weight / brdf_pdf;
        }
        else if (!mis && brdf_pdf > 0.f) {
            color += brdf * srd.emit * owl::abs(wi_local.z) / brdf_pdf;
        }
    }

    return color;
}

__device__
owl::common::vec3f sampleBRDFNoVis(SurfaceInteraction si, float lightSelectionPdf, owl::common::vec2f rand, bool mis)
{
    owl::common::vec3f wi_local = sample_GGX(rand, si.alpha, si.wo_local);
    owl::common::vec3f wi = normalize(apply_mat(si.to_world, wi_local));

    ShadowRay ray;
    ShadowRayData srd;
    ray.origin = si.p + 1e-3f * si.n_geom;
    ray.direction = wi;
    owl::traceRay(optixLaunchParams.world, ray, srd);

    owl::common::vec3f color(0.f, 0.f, 0.f);
    float light_pdf = 0.f, brdf_pdf = 0.f;

    if (true) {
        float xmy = pow(owl::length(srd.point - si.p), 2.f);
        float lDotWi = owl::abs(owl::dot(srd.normal, -wi));
        light_pdf = lightSelectionPdf * (xmy / (srd.area * lDotWi));

        owl::common::vec3f brdf = evaluate_brdf(si.wo_local, wi_local, si.diffuse, si.alpha);
        brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + wi_local));

        if (mis && light_pdf > 0.f && brdf_pdf > 0.f) {
            float weight = PowerHeuristic(1, brdf_pdf, 1, light_pdf);
            color += brdf * srd.emit * owl::abs(wi_local.z) * weight / brdf_pdf;
        }
        else if (!mis && brdf_pdf > 0.f) {
            color += brdf * srd.emit * owl::abs(wi_local.z) / brdf_pdf;
        }
    }

    return color;
}

__device__
owl::common::vec3f estimateDirectLighting(SurfaceInteraction& si, LCGRand& rng, int type)
{
    owl::common::vec2f rand1 = owl::common::vec2f(lcg_randomf(rng), lcg_randomf(rng));
    owl::common::vec2f rand2 = owl::common::vec2f(lcg_randomf(rng), lcg_randomf(rng));

    owl::common::vec3f lightSample = owl::common::vec3f(0.f);
    owl::common::vec3f brdfSample = owl::common::vec3f(0.f);
    owl::common::vec3f color = owl::common::vec3f(0.f);

    if (type == 0) {
        int selectedTriLight = lcg_randomf(rng) * (optixLaunchParams.numTriLights - 1);
        float lightSelectionPdf = 1.f / optixLaunchParams.numTriLights;

        lightSample = sampleLightSource(si, selectedTriLight, lightSelectionPdf, rand1, false);

        color = lightSample;
    }
    else if (type == 1) {
        brdfSample = sampleBRDF(si, 0.f, rand2, false);

        color = brdfSample;
    }
    else if (type == 2) {
        int selectedTriLight = lcg_randomf(rng) * (optixLaunchParams.numTriLights - 1);
        float lightSelectionPdf = 1.f / optixLaunchParams.numTriLights;

        brdfSample = sampleBRDF(si, lightSelectionPdf, rand1, true);
        lightSample = sampleLightSource(si, selectedTriLight, lightSelectionPdf, rand2, true);

        color = brdfSample + lightSample;
    }
    else if (type == 3) {
        int selectedTriLight = lcg_randomf(rng) * (optixLaunchParams.numTriLights - 1);
        float lightSelectionPdf = 1.f / optixLaunchParams.numTriLights;

        brdfSample = sampleBRDFNoNLTest(si, lightSelectionPdf, rand1, true);
        lightSample = sampleLightSourceNoNLTest(si, selectedTriLight, lightSelectionPdf, rand2, true);

        color = brdfSample + lightSample;
    }
    else if (type == 4) {
        int selectedTriLight = lcg_randomf(rng) * (optixLaunchParams.numTriLights - 1);
        float lightSelectionPdf = 1.f / optixLaunchParams.numTriLights;

        brdfSample = sampleBRDFNoVis(si, lightSelectionPdf, rand1, true);
        lightSample = sampleLightSourceNoVis(si, selectedTriLight, lightSelectionPdf, rand2, true);

        color = brdfSample + lightSample;
    }

    // Make sure there are no negative colors!
    color.x = owl::max(0.f, color.x);
    color.y = owl::max(0.f, color.y);
    color.z = owl::max(0.f, color.z);

    return color;
}

__device__
owl::common::vec3f estimatePathTracing(SurfaceInteraction& si, LCGRand& rng)
{
    owl::common::vec3f color = owl::common::vec3f(0.f);
    SurfaceInteraction _dummy_si = si;
    if (si.isLight)
        return si.emit;
    owl::common::vec3f tp(1., 1., 1.);
    for (int ray_depth = 0; ray_depth < 8; ray_depth++)
    {   
        owl::common::vec2f rand1 = owl::common::vec2f(lcg_randomf(rng), lcg_randomf(rng));
        owl::common::vec2f rand2 = owl::common::vec2f(lcg_randomf(rng), lcg_randomf(rng));

        int selectedTriLight = lcg_randomf(rng) * (optixLaunchParams.numTriLights - 1);
        float lightSelectionPdf = 1.f / optixLaunchParams.numTriLights;

        owl::common::vec3f light_sample(0.f, 0.f, 0.f), brdf_sample(0., 0., 0.);
        owl::common::vec2i pixel_id = owl::getLaunchIndex();
        bool mis = false; //  CHECk
        // Light sampling
        //{
        //    float light_pdf = 0.f, brdf_pdf = 0.f;
        //    TriLight triLight = optixLaunchParams.triLights[selectedTriLight];

        //    owl::common::vec3f lv1 = triLight.v1;
        //    owl::common::vec3f lv2 = triLight.v2;
        //    owl::common::vec3f lv3 = triLight.v3;
        //    owl::common::vec3f lnormal = triLight.normal;
        //    owl::common::vec3f lemit = triLight.emit;
        //    float larea = triLight.area;

        //    owl::common::vec3f lpoint = samplePointOnTriangle(lv1, lv2, lv3, rand1.x, rand1.y);
        //    owl::common::vec3f wi = normalize(lpoint - si.p);
        //    owl::common::vec3f wi_local = normalize(apply_mat(si.to_local, wi));

        //    float xmy = pow(owl::length(lpoint - si.p), 2.f);
        //    float lDotWi = owl::abs(owl::dot(lnormal, -wi));

        //    light_pdf = lightSelectionPdf * (xmy / (larea * lDotWi));

        //    ShadowRay ray;
        //    ray.origin = si.p + 1e-3f * si.n_geom;
        //    ray.direction = wi;
        //    //if(pixel_id.x == 1024 && pixel_id.y == 1024)
        //    //    printf("light %f, %f, %f\n", si.p.x, si.p.x, si.p.x);
        //    ShadowRayData srd;
        //    owl::traceRay(optixLaunchParams.world, ray, srd);

        //    if (si.wo_local.z > 0.f && wi_local.z > 0.f && srd.visibility != owl::common::vec3f(0.f) && light_pdf > 0.f && owl::dot(-wi, lnormal) > 0.f) {
        //        owl::common::vec3f brdf = evaluate_brdf(si.wo_local, wi_local, si.diffuse, si.alpha);
        //        brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + wi_local));

        //        if (mis && brdf_pdf > 0.f) {
        //            float weight = PowerHeuristic(1, light_pdf, 1, brdf_pdf);
        //            color += tp * brdf * lemit * owl::abs(wi_local.z) * weight / light_pdf;
        //        }
        //        else if (!mis) {
        //            color += tp * brdf * lemit * owl::abs(wi_local.z) / light_pdf;
        //        }
        //    }
        //    //light_sample *= owl::abs(owl::dot(owl::normalize(si.n_geom), owl::normalize(srd.point - si.p)));
        //}
        // BRDF sampling
        float lidotN = 0.;
        float lDotWi = 0.;
        SurfaceInteraction _si;
        {


            owl::common::vec3f wi_local = sample_GGX(rand2, si.alpha, si.wo_local);
            owl::common::vec3f wi = normalize(apply_mat(si.to_world, wi_local));

            ShadowRayData srd;

            ShadowRay shadow_ray;
            RadianceRay rad_ray;
            rad_ray.origin = si.p + 1e-3f * si.n_geom;
            rad_ray.direction = wi;

            /*if (pixel_id.x == 1024 && pixel_id.y == 1024)
                printf("brdf %f, %f, %f\n", si.p.x, si.p.x, si.p.x);
            */
            shadow_ray.origin = si.p + 1e-3f * si.n_geom;
            shadow_ray.direction = wi;
            owl::traceRay(optixLaunchParams.world, rad_ray, _si);
            owl::traceRay(optixLaunchParams.world, shadow_ray, srd);

            float light_pdf = 0.f, brdf_pdf = 0.f;
            
            lDotWi = owl::abs(owl::dot(_si.n_geom, -wi));
            lidotN = max(1e-6, owl::dot(_si.n_geom, -wi));
            owl::common::vec3f brdf = evaluate_brdf(si.wo_local, wi_local, si.diffuse, si.alpha);
            brdf_pdf = get_brdf_pdf(si.alpha, si.wo_local, normalize(si.wo_local + wi_local));
            //return owl::common::vec3f(lDotWi);
            if (wi_local.z > 0.f && si.wo_local.z > 0.f && srd.visibility != owl::common::vec3f(0.f)) {
                float xmy = pow(owl::length(srd.point - si.p), 2.f);

                light_pdf = lightSelectionPdf * (xmy / (srd.area * lDotWi));

                if (mis && light_pdf > 0.f && brdf_pdf > 0.f) {
                    float weight = PowerHeuristic(1, brdf_pdf, 1, light_pdf);
                    color += tp * brdf * srd.emit * owl::abs(wi_local.z) * weight / brdf_pdf;
                }
                else if (!mis && brdf_pdf > 0.f) {
                    color += tp * brdf * srd.emit * owl::abs(wi_local.z) / brdf_pdf;
                }
                break;
            }
            tp *= lidotN * brdf / brdf_pdf;

            if (!_si.hit)
                break;
            si = _si;
        }
    }
    si = _dummy_si;
    return color;
}


__device__
owl::common::vec3f ltcDirectLighingBaseline(SurfaceInteraction& si, LCGRand& rng)
{
    owl::common::vec3f wo_local = normalize(apply_mat(si.to_local, si.wo));
    if (wo_local.z < 0.f)
        return owl::common::vec3f(0.f);

    owl::common::vec3f normal_local(0.f, 0.f, 1.f);
    owl::common::vec3f color(0.0, 0.0, 0.0);

    /* Analytic shading via LTCs */
    owl::common::vec3f ltc_mat[3], ltc_mat_inv[3];
    float alpha = si.alpha;
    float theta = sphericalTheta(wo_local);

    float amplitude = 1.f;
    fetchLtcMat(alpha, theta, ltc_mat, amplitude);
    matrixInverse(ltc_mat, ltc_mat_inv);

    owl::common::vec3f iso_frame[3];

    iso_frame[0] = wo_local;
    iso_frame[0].z = 0.f;
    iso_frame[0] = normalize(iso_frame[0]);
    iso_frame[2] = normal_local;
    iso_frame[1] = normalize(owl::cross(iso_frame[2], iso_frame[0]));

    for (int lidx = 0; lidx < optixLaunchParams.numTriLights; lidx++) {
        color += integrateOverPolygon(si, ltc_mat, ltc_mat_inv, amplitude, iso_frame,
            optixLaunchParams.triLights[lidx]);
    }

    return color;
}


OPTIX_RAYGEN_PROGRAM(rayGen)()
{
    const RayGenData& self = owl::getProgramData<RayGenData>();
    const owl::common::vec2i pixelId = owl::getLaunchIndex();
    const int fbOfs = pixelId.x + self.frameBufferSize.x * pixelId.y;

    LCGRand rng = get_rng(optixLaunchParams.accumId + 10007, make_uint2(pixelId.x, pixelId.y),
        make_uint2(self.frameBufferSize.x, self.frameBufferSize.y));

    const owl::common::vec2f screen = (owl::common::vec2f(pixelId) + owl::common::vec2f(lcg_randomf(rng), lcg_randomf(rng))) / owl::common::vec2f(self.frameBufferSize);
    RadianceRay ray;
    ray.origin
        = optixLaunchParams.camera.pos;
    ray.direction
        = normalize(optixLaunchParams.camera.dir_00
            + screen.u * optixLaunchParams.camera.dir_du
            + screen.v * optixLaunchParams.camera.dir_dv);

    SurfaceInteraction si;
    owl::traceRay(optixLaunchParams.world, ray, si);

    owl::common::vec3f color(0.f, 0.f, 0.f);
    //printf("%d\n", optixLaunchParams.rendererType);
    if (si.hit == false)
    {
        color = si.diffuse;
        color = si.n_geom;
    }
    else if (optixLaunchParams.rendererType == MASK)
        color = owl::common::vec3f(1., 1., 1.);
    else if (optixLaunchParams.rendererType == POSITION)
        color = si.p;
    else if (optixLaunchParams.rendererType == DIFFUSE)
        color = si.diffuse;
    else if (optixLaunchParams.rendererType == ALPHA)
        color = si.alpha;
    else if (optixLaunchParams.rendererType == NORMALS)
        color = si.n_geom;
    // Direct lighting with MC
    else if (optixLaunchParams.rendererType == DIRECT_LIGHT_LSAMPLE) {
        if (si.isLight)
            color = si.emit;
        else
            color = estimateDirectLighting(si, rng, 0);
    }
    else if (optixLaunchParams.rendererType == DIRECT_LIGHT_BRDFSAMPLE) {
        if (si.isLight)
            color = si.emit;
        else
            color = estimateDirectLighting(si, rng, 1);
    }
    else if (optixLaunchParams.rendererType == DIRECT_LIGHT_MIS) {
        if (si.isLight)
            color = si.emit;
        else
            color = estimateDirectLighting(si, rng, 2);
    }
    // Direct lighting with LTC
    else if (optixLaunchParams.rendererType == LTC_BASELINE) {
        if (si.isLight)
            color = si.emit;
        else
            color = ltcDirectLighingBaseline(si, rng);
    }
    else if (optixLaunchParams.rendererType == RATIO) {
        owl::common::vec3f ltc_color = owl::common::vec3f(0.);
        owl::common::vec3f sto_S = owl::common::vec3f(0.);
        owl::common::vec3f sto_U = owl::common::vec3f(0.);

        if (si.isLight)
            color = si.emit;
        else {
            ltc_color = ltcDirectLighingBaseline(si, rng);
            for (int i = 0; i < 4; i++)
            {
                sto_S += estimateDirectLighting(si, rng, 3);
                sto_U += estimateDirectLighting(si, rng, 4);
            }

            color.x = (sto_U.x < 1e-4) ? 0. : ltc_color.x * sto_S.x / sto_U.x;
            color.y = (sto_U.y < 1e-4) ? 0. : ltc_color.y * sto_S.y / sto_U.y;
            color.z = (sto_U.z < 1e-4) ? 0. : ltc_color.z * sto_S.z / sto_U.z;
            //color = ltc_color * sto_S / sto_U;
        }

    }
    else if (optixLaunchParams.rendererType == PATH)
    {
        int n = 1;
        for(int i=0;i<n;i++)
           color += estimatePathTracing(si, rng);
        color /= n;
    }
    else {
        color = owl::common::vec3f(1., 0., 0.);
    }

    if (optixLaunchParams.accumId > 0)
        color = color + owl::common::vec3f(optixLaunchParams.accumBuffer[fbOfs].x, optixLaunchParams.accumBuffer[fbOfs].y,
            optixLaunchParams.accumBuffer[fbOfs].z);

    optixLaunchParams.accumBuffer[fbOfs] = make_float4(color.x, color.y, color.z, 1.f);
    color = (1.f / (optixLaunchParams.accumId + 1)) * color;
    self.frameBuffer[fbOfs] = owl::make_rgba(color);

    //self.frameBuffer[fbOfs] = owl::make_rgba(owl::common::vec3f(0., 1., 0.));
}
